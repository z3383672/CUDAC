
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void vecAdd(float* A, float* B, float* C, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int n = 100000;
    size_t bytes = n * sizeof(float);

    // Allocate memory on the host
    std::vector<float> h_A(n, 1.1f);
    std::vector<float> h_B(n, 2.2f);
    std::vector<float> h_C(n);

    // Allocate memory on the device
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from the host to the device
    hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice);

    // Set up the kernel launch parameters
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch the kernel
    vecAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);

    // Copy data from the device to the host
    hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Print the results
    for (int i = 0; i < n; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
